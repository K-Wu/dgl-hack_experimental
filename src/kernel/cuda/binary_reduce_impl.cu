#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_impl.cu
 * \brief Binary reduce implementation on cuda.
 */
#include <algorithm>
#include <hip/hip_runtime.h>

#include "../binary_reduce_impl.h"
#include "../csr_interface.h"

using dgl::runtime::NDArray;

namespace dgl {
namespace kernel {

template <typename DType>
__device__ DType gatLeakyReluExp(DType val, DType slope) {
    return val > 0 ? exp(val) : exp(slope * val);
}

template <typename Idx, typename DType>
__global__ void gatExpLeakyReluSumKernel(GatFusedData<Idx, DType> gdata, minigun::Csr<Idx> csr) {
    Idx tx = blockIdx.x * blockDim.x + threadIdx.x;
    Idx ty = blockIdx.y * blockDim.y + threadIdx.y;
    Idx stride_x = blockDim.x * gridDim.x;
    Idx stride_y = blockDim.y * gridDim.y;
    Idx feat_idx = tx;
    Idx dst_vid = ty;
    DType e_xlen = gdata.e_xlen;
    while (dst_vid < csr.row_offsets.length) {
        Idx start_off = *(csr.row_offsets.data + dst_vid);
        Idx end_off = *(csr.row_offsets.data + dst_vid + 1);
        while (feat_idx < e_xlen) {
            DType sum = 0.;
            for (Idx eid=start_off; eid<end_off; ++eid) {
                Idx src_id = *(csr.column_indices.data + eid);
                Idx feat_off_src = src_id * e_xlen + feat_idx;
                Idx feat_off_dst = dst_vid * e_xlen + feat_idx;
                DType tmp = gatLeakyReluExp(gdata.el[feat_off_src] + gdata.er[feat_off_dst], gdata.leaky_relu_slope);
                gdata.exp[Idx(eid * e_xlen) + feat_idx] = tmp;
                sum += tmp;
            }
            gdata.sum[Idx(dst_vid*e_xlen) + feat_idx] = sum;
            feat_idx += stride_x;
        }
        dst_vid += stride_y;
    }
}

template <typename Idx, typename DType>
__global__ void gatSumProdZipDivKernel(GatFusedData<Idx, DType> gdata, minigun::Csr<Idx> csr) {
    Idx tx = blockIdx.x * blockDim.x + threadIdx.x;
    Idx ty = blockIdx.y * blockDim.y + threadIdx.y;
    Idx stride_x = blockDim.x * gridDim.x;
    Idx stride_y = blockDim.y * gridDim.y;
    DType e_xlen = gdata.e_xlen;
    DType feat_src_xlen = gdata.feat_src_xlen;
    Idx dst_vid = ty;
    while (dst_vid < csr.row_offsets.length) {
        Idx start_off = *(csr.row_offsets.data + dst_vid);
        Idx end_off = *(csr.row_offsets.data + dst_vid + 1);
        Idx head_offset = tx;
        while (head_offset < e_xlen) {
            DType ret = 0.;
            Idx hidden_offset = threadIdx.y;
            while (hidden_offset < gdata.feat_src_hidden) {
                for (Idx eid=start_off; eid<end_off; ++eid) {
                    Idx src_id = *(csr.column_indices + eid);
                    DType ex = *(gdata.exp + src_id*e_xlen + head_offset);
                    DType s = *(gdata.sum + dst_vid * e_xlen + head_offset);
                    DType feat_src = *(gdata.feat_src + src_id * feat_src_xlen + head_offset * gdata.feat_src_hidden + hidden_offset);
                    ret += ex/s*feat_src;
                }
                gdata.ret[gdata.feat_src + src_id * feat_src_xlen + head_offset * gdata.feat_src_hidden + hidden_offset] = ret;
                hidden_offset += blockDim.y;
            }
            head_offset += stride_x;
        }
        dst_vid += stride_y;
    }
}

void FusedGatKernelImpl(
    const CSRWrapper& graph,
    runtime::NDArray feat_src,
    runtime::NDArray el,
    runtime::NDArray er,
    runtime::NDArray sum,
    runtime::NDArray exp,
    runtime::NDArray ret,
    float slope) {
        typedef int32_t Idx;
        typedef float DType;
        const Idx MAX_NBLKS = 65535;
        // zero out ret, and packing feat_src, el, er, ret, graph together into one struct using raw float pointers
        // get csr matrix
        GatFusedData<Idx, DType> gdata;
        int64_t el_xlen =  utils::ComputeXLength(el);
        int64_t feat_src_xlen =  utils::ComputeXLength(feat_src);
        int64_t ret_len =  utils::ComputeXLength(ret);
        gdata.feat_src = static_cast<DType*>(feat_src->data);
        gdata.el = static_cast<DType*>(el->data);
        gdata.er = static_cast<DType*>(er->data);
        gdata.sum = static_cast<DType*>(sum->data);
        gdata.exp = static_cast<DType*>(exp->data);
        gdata.ret = static_cast<DType*>(ret->data);
        gdata.leaky_relu_slope = slope;
        gdata.n = el.GetSize()/sizeof(DType)/el_xlen; 
        gdata.e_xlen = el_xlen;
        gdata.feat_src_xlen =  feat_src_xlen;
        gdata.feat_src_hidden = feat_src_xlen/el_xlen;
        gdata.ret_xlen = ret_len;
        auto incsr = graph.GetInCSRMatrix();
        minigun::Csr<Idx> csr = utils::CreateCsr<Idx>(incsr.indptr, incsr.indices);
        // write a device function and call it from here
        LOG(INFO) << "Within Fused Gat Kernel Impl." << "feat_src_dim:" << feat_src.GetSize()/sizeof(DType)/feat_src_xlen << "*" << feat_src_xlen 
            <<" el_dim:" << el.GetSize()/sizeof(DType)/el_xlen << "*" << el_xlen  << " ret_dim:" << ret.GetSize()/sizeof(DType)/ret_len <<"*" << ret_len
            << " graph csr row_offset length:" <<csr.row_offsets.length << " graph csr column indices length:" << csr.column_indices.length;

        // Configure kernel launch parameters.
        auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
        int max_xlen = std::max(feat_src_xlen, el_xlen);
        int nthrs_x = utils::FindNumThreads(max_xlen, 64);
        int nthrs_y = 1;
        int nblks_x = (max_xlen + nthrs_x-1)/(nthrs_x);
        int nblks_y = std::min((gdata.n + nthrs_y -1)/nthrs_y, MAX_NBLKS);
        const dim3 nblks(nblks_x, nblks_y);
        const dim3 nthrs(nthrs_x, nthrs_y);
        LOG(INFO) << "blk dim:" << nblks_x << "*" <<nblks_y << " nthrs:" <<nthrs_x << "*" << nthrs_y;
        gatExpLeakyReluSumKernel<<<nblks, nthrs, 0, thr_entry->stream>>>(gdata, csr);

        nthrs_x = feat_src_xlen / gdata.feat_src_hidden;
        nthrs_y = gdata.feat_src_hidden;
        nblks_x = 1;
        nblks_y = std::min((gdata.n + nthrs_y -1)/nthrs_y, MAX_NBLKS);
        const dim3 nblks2(nblks_x, nblks_y);
        const dim3 nthrs2(nthrs_x, nthrs_y);
        gatSumProdZipDivKernel<<<nblks, nthrs, 0, thr_entry->stream>>>(gdata, csr);
    }

template void BinaryReduceImpl<kDLGPU>(
    const std::string& reducer,
    const std::string& op,
    const CSRWrapper& graph,
    binary_op::Target lhs, binary_op::Target rhs,
    runtime::NDArray lhs_data, runtime::NDArray rhs_data,
    runtime::NDArray out_data,
    runtime::NDArray lhs_mapping, runtime::NDArray rhs_mapping,
    runtime::NDArray out_mapping);

template void BinaryReduceBcastImpl<kDLGPU>(
    const BcastInfo& info,
    const std::string& reducer,
    const std::string& op,
    const CSRWrapper& graph,
    binary_op::Target lhs, binary_op::Target rhs,
    runtime::NDArray lhs_data, runtime::NDArray rhs_data,
    runtime::NDArray out_data,
    runtime::NDArray lhs_mapping, runtime::NDArray rhs_mapping,
    runtime::NDArray out_mapping);

template void BackwardBinaryReduceImpl<kDLGPU>(
    const std::string& reducer,
    const std::string& op,
    const CSRWrapper& graph,
    binary_op::Target lhs, binary_op::Target rhs,
    NDArray lhs_mapping, NDArray rhs_mapping, NDArray out_mapping,
    NDArray lhs_data, NDArray rhs_data, NDArray out_data,
    NDArray grad_out_data,
    NDArray grad_lhs_data, NDArray grad_rhs_data);

template void BackwardBinaryReduceBcastImpl<kDLGPU>(
    const BcastInfo& info,
    const std::string& reducer,
    const std::string& op,
    const CSRWrapper& graph,
    binary_op::Target lhs_tgt, binary_op::Target rhs_tgt,
    runtime::NDArray lhs_mapping, runtime::NDArray rhs_mapping, runtime::NDArray out_mapping,
    runtime::NDArray lhs, runtime::NDArray rhs, runtime::NDArray out, runtime::NDArray grad_out,
    runtime::NDArray grad_lhs, runtime::NDArray grad_rhs);

}  // namespace kernel
}  // namespace dgl
